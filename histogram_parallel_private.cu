#include "hip/hip_runtime.h"
__global__ void parallel_private_histogram_kernel(char *data, unsigned int length, unsigned int *histo) {
    __shared__ unsigned int histo_s[NUM_BINS]
    for (unsigned int bin = threadIdx.x; bin < NUM_BINS; bin += blockDim.x) {
        histo_s[bin] = 0u;
    }

    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < length) {
        int alphabet_pos = data[i] - 'a';
        if (alphabet_pos >= 0 && alphabet_pos < 26) {
            atomicAdd(&(histo_s[blockIdx.x*NUM_BINS + alphabet_pos/4]), 1);
        }
    }
    if (blockIdx.x > 0) {
        __syncthreads();
        for (unsigned int bin = threadIdx.x; bin < NUM_BINS; bin += blockDim.x) {
            unsigned int binValue = histo_s[blockIdx.x*NUM_BINS + bin];
            if (binValue > 0) {
                atmoicAdd(&(histo[bin]), binValue);
            }
        }
    }
}