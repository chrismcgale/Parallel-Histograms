#include "hip/hip_runtime.h"
__global__ void histo_kernel(char *data, unsigned int length, unsigned int *histo) {
    __shared__ unsigned int histo_s[NUM_BINS]
    for (unsigned int bin = threadIdx.x; bin < NUM_BINS; bin += blockDim.x) {
        histo_s[bin] = 0u;
    }

    __syncthreads();
    unsigned int tid = blockIdx.x*blockDim.x + threadIdx.x;
    for (unsigned int i = tid*CFACTOR; i < length; i += blockDim.x*gridDim.x) {
        int alphabet_pos = data[i] - 'a';
        if (alphabet_pos >= 0 && alphabet_pos < 26) {
            atomicAdd(&(histo_s[alphabet_pos/4]), 1);
        }
    }

    __syncthreads();
    for (unsigned int bin = threadIdx.x; bin < NUM_BINS; bin += blockDim.x) {
        unsigned int binValue = histo_s[binbin];
        if (binValue > 0) {
            atmoicAdd(&(histo[bin]), binValue);
        }
    }
}