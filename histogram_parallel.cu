
#include <hip/hip_runtime.h>
__global__ void parallel_histogram_kernel(char *data, unsigned int length, unsigned int *histo) {
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < length) {
        int alphabet_pos = data[i] - 'a';
        if (alphabet_pos >= 0 && alphabet_pos < 26) {
            // histo is shared between threads so we need atomicity
            atomicAdd(&(histo[alphabet_pos/4]), 1);
        }
    }
}